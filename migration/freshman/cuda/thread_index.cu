#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}

void printMatrix(float * C,const int nx,const int ny)
{
  float *ic=C;
  printf("Matrix<%d,%d>:\n",ny,nx);
  for(int i=0;i<ny;i++)
  {
    for(int j=0;j<nx;j++)
    {
      printf("%6f ",ic[j]);
    }
    ic+=nx;
    printf("\n");
  }
}

void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp,dev));
  printf("Using device %d: %s\n",dev,deviceProp.name);
  CHECK(hipSetDevice(dev));

}

void initialData(float* ip,int size)
{
  time_t t;
  srand((unsigned )time(&t));
  for(int i=0;i<size;i++)
  {
    ip[i]=(float)(rand()&0xffff)/1000.0f;
  }
}



__global__ void printThreadIndex(float *A,const int nx,const int ny)
{
  int ix=threadIdx.x+blockIdx.x*blockDim.x;
  int iy=threadIdx.y+blockIdx.y*blockDim.y;
  unsigned int idx=iy*nx+ix;
  printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d)"
          "global index %2d ival %f\n",threadIdx.x,threadIdx.y,
          blockIdx.x,blockIdx.y,ix,iy,idx,A[idx]);
}
int main(int argc,char** argv)
{
  initDevice(0);
  int nx=8,ny=6;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  printMatrix(A_host,nx,ny);

  //hipMalloc
  float *A_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));

  hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice);

  dim3 block(4,2);
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);

  printThreadIndex<<<grid,block>>>(A_dev,nx,ny);

  CHECK(hipDeviceSynchronize());
  hipFree(A_dev);
  free(A_host);

  hipDeviceReset();
  return 0;
}
