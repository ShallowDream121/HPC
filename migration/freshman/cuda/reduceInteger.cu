#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}
double cpuSecond()
{
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return((double)tp.tv_sec+(double)tp.tv_usec*1e-6);

}
void initialData_int(int* ip, int size)
{
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i<size; i++)
	{
		ip[i] = int(rand()&0xff);
	}
}
void initDevice(int devNum)
{
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp,dev));
  printf("Using device %d: %s\n",dev,deviceProp.name);
  CHECK(hipSetDevice(dev));

}
int recursiveReduce(int *data, int const size)
{
	if (size == 1) return data[0];
	int const stride = size / 2;
	if (size % 2 == 1)
	{
		for (int i = 0; i < stride; i++)
		{
			data[i] += data[i + stride];
		}
		data[0] += data[size - 1];
	}
	else
	{
		for (int i = 0; i < stride; i++)
		{
			data[i] += data[i + stride];
		}
	}
	return recursiveReduce(data, stride);
}
__global__ void warmup(int * g_idata, int * g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	if (tid >= n) return;
	int *idata = g_idata + blockIdx.x*blockDim.x;
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}
__global__ void reduceNeighbored(int * g_idata,int * g_odata,unsigned int n) 
{
	unsigned int tid = threadIdx.x;
	if (tid >= n) return;
	int *idata = g_idata + blockIdx.x*blockDim.x;
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];

}

__global__ void reduceNeighboredLess(int * g_idata,int *g_odata,unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	int *idata = g_idata + blockIdx.x*blockDim.x;
	if (idx > n)
		return;
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		int index = 2 * stride *tid;
		if (index < blockDim.x)
		{
			idata[index] += idata[index + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceInterleaved(int * g_idata, int *g_odata, unsigned int n)
{
	unsigned int tid = threadIdx.x;
	unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
	int *idata = g_idata + blockIdx.x*blockDim.x;
	if (idx >= n)
		return;
	for (int stride = blockDim.x/2; stride >0; stride >>=1)
	{
		
		if (tid <stride)
		{
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0)
		g_odata[blockIdx.x] = idata[0];
}
int main(int argc,char** argv)
{
	initDevice(0);
	
	bool bResult = false;

	int size = 1 << 24;
	printf("	with array size %d  ", size);

	int blocksize = 1024;
	if (argc > 1)
	{
		blocksize = atoi(argv[1]);
	}
	dim3 block(blocksize, 1);
	dim3 grid((size - 1) / block.x + 1, 1);
	printf("grid %d block %d \n", grid.x, block.x);

	size_t bytes = size * sizeof(int);
	int *idata_host = (int*)malloc(bytes);
	int *odata_host = (int*)malloc(grid.x * sizeof(int));
	int * tmp = (int*)malloc(bytes);

	initialData_int(idata_host, size);

	memcpy(tmp, idata_host, bytes);
	double iStart, iElaps;
	int gpu_sum = 0;

	int * idata_dev = NULL;
	int * odata_dev = NULL;
	CHECK(hipMalloc((void**)&idata_dev, bytes));
	CHECK(hipMalloc((void**)&odata_dev, grid.x * sizeof(int)));

	int cpu_sum = 0;
	iStart = cpuSecond();
	for (int i = 0; i < size; i++)
		cpu_sum += tmp[i];
	printf("cpu sum:%d \n", cpu_sum);
	iElaps = cpuSecond() - iStart;
	printf("cpu reduce                 elapsed %lf ms cpu_sum: %d\n", iElaps, cpu_sum);

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	warmup <<<grid, block >>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu warmup                 elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	reduceNeighbored << <grid, block >> >(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu reduceNeighbored       elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	reduceNeighboredLess <<<grid, block>>>(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu reduceNeighboredLess   elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);

	CHECK(hipMemcpy(idata_dev, idata_host, bytes, hipMemcpyHostToDevice));
	CHECK(hipDeviceSynchronize());
	iStart = cpuSecond();
	reduceInterleaved << <grid, block >> >(idata_dev, odata_dev, size);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	hipMemcpy(odata_host, odata_dev, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_sum += odata_host[i];
	printf("gpu reduceInterleaved      elapsed %lf ms gpu_sum: %d<<<grid %d block %d>>>\n",
		iElaps, gpu_sum, grid.x, block.x);
	free(idata_host);
	free(odata_host);
	CHECK(hipFree(idata_dev));
	CHECK(hipFree(odata_dev));

	hipDeviceReset();

	if (gpu_sum == cpu_sum)
	{
		printf("Test success!\n");
	}
	return EXIT_SUCCESS;
}
