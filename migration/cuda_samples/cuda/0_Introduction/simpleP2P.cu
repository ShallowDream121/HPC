#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#ifndef EXIT_WAIVED
#define EXIT_WAIVED 2
#endif

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

__global__ void SimpleKernel(float *src, float *dst) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx] * 2.0f;
}

inline bool IsAppBuiltAs64() { return sizeof(void *) == 8; }

int main(int argc, char **argv) {
  printf("[%s] - Starting...\n", argv[0]);

  if (!IsAppBuiltAs64()) {
    printf(
        "%s is only supported with on 64-bit OSs and the application must be "
        "built as a 64-bit target.  Test is being waived.\n",
        argv[0]);
    exit(EXIT_WAIVED);
  }
  printf("Checking for multiple GPUs...\n");
  int gpu_n;
  checkCudaErrors(hipGetDeviceCount(&gpu_n));
  printf("CUDA-capable device count: %i\n", gpu_n);

  if (gpu_n < 2) {
    printf(
        "Two or more GPUs with Peer-to-Peer access capability are required for "
        "%s.\n",
        argv[0]);
    printf("Waiving test.\n");
    exit(EXIT_WAIVED);
  }

  hipDeviceProp_t prop[64];
  int gpuid[2]; 

  for (int i = 0; i < gpu_n; i++) {
    checkCudaErrors(hipGetDeviceProperties(&prop[i], i));
  }
  printf("\nChecking GPU(s) for support of peer to peer memory access...\n");

  int can_access_peer;
  int p2pCapableGPUs[2]; 
  p2pCapableGPUs[0] = p2pCapableGPUs[1] = -1;

  for (int i = 0; i < gpu_n; i++) {
    for (int j = 0; j < gpu_n; j++) {
      if (i == j) {
        continue;
      }
      checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer, i, j));
      printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[i].name,
             i, prop[j].name, j, can_access_peer ? "Yes" : "No");
      if (can_access_peer && p2pCapableGPUs[0] == -1) {
        p2pCapableGPUs[0] = i;
        p2pCapableGPUs[1] = j;
      }
    }
  }

  if (p2pCapableGPUs[0] == -1 || p2pCapableGPUs[1] == -1) {
    printf(
        "Two or more GPUs with Peer-to-Peer access capability are required for "
        "%s.\n",
        argv[0]);
    printf(
        "Peer to Peer access is not available amongst GPUs in the system, "
        "waiving test.\n");

    exit(EXIT_WAIVED);
  }

  gpuid[0] = p2pCapableGPUs[0];
  gpuid[1] = p2pCapableGPUs[1];

  printf("Enabling peer access between GPU%d and GPU%d...\n", gpuid[0],
         gpuid[1]);
  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[1], 0));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[0], 0));

  const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
  printf("Allocating buffers (%iMB on GPU%d, GPU%d and CPU Host)...\n",
         int(buf_size / 1024 / 1024), gpuid[0], gpuid[1]);
  checkCudaErrors(hipSetDevice(gpuid[0]));
  float *g0;
  checkCudaErrors(hipMalloc(&g0, buf_size));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  float *g1;
  checkCudaErrors(hipMalloc(&g1, buf_size));
  float *h0;
  checkCudaErrors(
      hipHostMalloc(&h0, buf_size, hipHostMallocDefault)); 

  printf("Creating event handles...\n");
  hipEvent_t start_event, stop_event;
  float time_memcpy;
  int eventflags = hipEventBlockingSync;
  checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
  checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

  checkCudaErrors(hipEventRecord(start_event, 0));

  for (int i = 0; i < 100; i++) {
    if (i % 2 == 0) {
      checkCudaErrors(hipMemcpy(g1, g0, buf_size, hipMemcpyDefault));
    } else {
      checkCudaErrors(hipMemcpy(g0, g1, buf_size, hipMemcpyDefault));
    }
  }

  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
  printf("hipMemcpyPeer / hipMemcpy between GPU%d and GPU%d: %.2fGB/s\n",
         gpuid[0], gpuid[1],
         (1.0f / (time_memcpy / 1000.0f)) * ((100.0f * buf_size)) / 1024.0f /
             1024.0f / 1024.0f);

  printf("Preparing host buffer and memcpy to GPU%d...\n", gpuid[0]);

  for (int i = 0; i < buf_size / sizeof(float); i++) {
    h0[i] = float(i % 4096);
  }

  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipMemcpy(g0, h0, buf_size, hipMemcpyDefault));

  const dim3 threads(512, 1);
  const dim3 blocks((buf_size / sizeof(float)) / threads.x, 1);

  printf(
      "Run kernel on GPU%d, taking source data from GPU%d and writing to "
      "GPU%d...\n",
      gpuid[1], gpuid[0], gpuid[1]);
  checkCudaErrors(hipSetDevice(gpuid[1]));
  SimpleKernel<<<blocks, threads>>>(g0, g1);

  checkCudaErrors(hipDeviceSynchronize());

  printf(
      "Run kernel on GPU%d, taking source data from GPU%d and writing to "
      "GPU%d...\n",
      gpuid[0], gpuid[1], gpuid[0]);
  checkCudaErrors(hipSetDevice(gpuid[0]));
  SimpleKernel<<<blocks, threads>>>(g1, g0);

  checkCudaErrors(hipDeviceSynchronize());

  printf("Copy data back to host from GPU%d and verify results...\n", gpuid[0]);
  checkCudaErrors(hipMemcpy(h0, g0, buf_size, hipMemcpyDefault));

  int error_count = 0;

  for (int i = 0; i < buf_size / sizeof(float); i++) {
    if (h0[i] != float(i % 4096) * 2.0f * 2.0f) {
      printf("Verification error @ element %i: val = %f, ref = %f\n", i, h0[i],
             (float(i % 4096) * 2.0f * 2.0f));

      if (error_count++ > 10) {
        break;
      }
    }
  }

  printf("Disabling peer access...\n");
  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[1]));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[0]));

  printf("Shutting down...\n");
  checkCudaErrors(hipEventDestroy(start_event));
  checkCudaErrors(hipEventDestroy(stop_event));
  checkCudaErrors(hipSetDevice(gpuid[0]));
  checkCudaErrors(hipFree(g0));
  checkCudaErrors(hipSetDevice(gpuid[1]));
  checkCudaErrors(hipFree(g1));
  checkCudaErrors(hipHostFree(h0));

  for (int i = 0; i < gpu_n; i++) {
    checkCudaErrors(hipSetDevice(i));
  }

  if (error_count != 0) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  } else {
    printf("Test passed\n");
    exit(EXIT_SUCCESS);
  }
}
