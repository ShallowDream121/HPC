#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

inline const char* _ConvertSMVer2ArchName(int major, int minor) {
  typedef struct {
    int SM; 
    const char* name;
  } sSMtoArchName;

  sSMtoArchName nGpuArchNameSM[] = {
      {0x30, "Kepler"},
      {0x32, "Kepler"},
      {0x35, "Kepler"},
      {0x37, "Kepler"},
      {0x50, "Maxwell"},
      {0x52, "Maxwell"},
      {0x53, "Maxwell"},
      {0x60, "Pascal"},
      {0x61, "Pascal"},
      {0x62, "Pascal"},
      {0x70, "Volta"},
      {0x72, "Xavier"},
      {0x75, "Turing"},
      {0x80, "Ampere"},
      {0x86, "Ampere"},
      {0x87, "Ampere"},
      {0x89, "Ada"},
      {0x90, "Hopper"},
      {0xa0, "Blackwell"},
      {0xa1, "Blackwell"},
      {0xc0, "Blackwell"},
      {-1, "Graphics Device"}};

  int index = 0;

  while (nGpuArchNameSM[index].SM != -1) {
    if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchNameSM[index].name;
    }

    index++;
  }

  printf(
      "MapSMtoArchName for SM %d.%d is undefined."
      "  Default to use %s\n",
      major, minor, nGpuArchNameSM[index - 1].name);
  return nGpuArchNameSM[index - 1].name;
}

inline int _ConvertSMVer2Cores(int major, int minor) {
  typedef struct {
    int SM;
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {0x87, 128},
      {0x89, 128},
      {0x90, 128},
      {0xa0, 128},
      {0xa1, 128},
      {0xc0, 128},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  printf(
      "MapSMtoCores for SM %d.%d is undefined."
      "  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

inline int gpuGetMaxGflopsDeviceId() {
  int current_device = 0, sm_per_multiproc = 0;
  int max_perf_device = 0;
  int device_count = 0;
  int devices_prohibited = 0;

  uint64_t max_compute_perf = 0;
  checkCudaErrors(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  current_device = 0;

  while (current_device < device_count) {
    int computeMode = -1, major = 0, minor = 0;
    checkCudaErrors(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, current_device));

    if (computeMode != hipComputeModeProhibited) {
      if (major == 9999 && minor == 9999) {
        sm_per_multiproc = 1;
      } else {
        sm_per_multiproc =
            _ConvertSMVer2Cores(major,  minor);
      }
      int multiProcessorCount = 0, clockRate = 0;
      checkCudaErrors(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, current_device));
      hipError_t result = hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, current_device);
      if (result != hipSuccess) {
        if(result == hipErrorInvalidValue) {
          clockRate = 1;
        }
        else {
          exit(EXIT_FAILURE);
        }
      }
      uint64_t compute_perf = (uint64_t)multiProcessorCount * sm_per_multiproc * clockRate;

      if (compute_perf > max_compute_perf) {
        max_compute_perf = compute_perf;
        max_perf_device = current_device;
      }
    } else {
      devices_prohibited++;
    }

    ++current_device;
  }

  if (devices_prohibited == device_count) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " all devices have compute mode prohibited.\n");
    exit(EXIT_FAILURE);
  }

  return max_perf_device;
}

inline int findCudaDevice(int argc, const char **argv) {
  int devID = 0;

  devID = gpuGetMaxGflopsDeviceId();
  checkCudaErrors(hipSetDevice(devID));
  int major = 0, minor = 0;
  checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
  checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
  printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
         devID, _ConvertSMVer2ArchName(major, minor), major, minor);

  return devID;
}

template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(float *C, float *A,
    float *B, int wA,
    int wB) {
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int aBegin = wA * BLOCK_SIZE * by;

  int aEnd   = aBegin + wA - 1;

  int aStep  = BLOCK_SIZE;

  int bBegin = BLOCK_SIZE * bx;

  int bStep  = BLOCK_SIZE * wB;

  float Csub = 0;

  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    __syncthreads();

#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    __syncthreads();
  }

  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

void ConstantInit(float *data, int size, float val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  hipStream_t stream;

  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  float *d_A, *d_B, *d_C;

  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(
      hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

  printf("Computing result using CUDA Kernel...\n");

  if (block_size == 16) {
    MatrixMulCUDA<16>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  } else {
    MatrixMulCUDA<32>
        <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
  }

  printf("done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  checkCudaErrors(hipEventRecord(start, stream));

  int nIter = 300;

  for (int j = 0; j < nIter; j++) {
    if (block_size == 16) {
      MatrixMulCUDA<16>
          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    } else {
      MatrixMulCUDA<32>
          <<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
  }

  checkCudaErrors(hipEventRecord(stop, stream));

  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  float msecPerMatrixMul = msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                             static_cast<double>(dimsA.y) *
                             static_cast<double>(dimsB.x);
  double gigaFlops =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf(
      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
      " WorkgroupSize= %u threads/block\n",
      gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

  checkCudaErrors(
      hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Checking computed result for correctness: ");
  bool correct = true;

  double eps = 1.e-6;

  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
             i, h_C[i], dimsA.x * valB, eps);
      correct = false;
    }
  }

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  printf(
      "\nNOTE: The CUDA Samples are not meant for performance "
      "measurements. Results may vary when GPU Boost is enabled.\n");

  if (correct) {
    return EXIT_SUCCESS;
  } else {
    return EXIT_FAILURE;
  }
}

int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
  dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  checkCudaErrors(hipProfilerStart());
  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);
  checkCudaErrors(hipProfilerStop());

  exit(matrix_result);
}
