#include <sys/utsname.h>

#include <stdio.h>
#include <stdint.h>
#include <cassert>

#include <hip/hip_runtime.h>

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

inline const char* _ConvertSMVer2ArchName(int major, int minor) {
  typedef struct {
    int SM; 
    const char* name;
  } sSMtoArchName;

  sSMtoArchName nGpuArchNameSM[] = {
      {0x30, "Kepler"},
      {0x32, "Kepler"},
      {0x35, "Kepler"},
      {0x37, "Kepler"},
      {0x50, "Maxwell"},
      {0x52, "Maxwell"},
      {0x53, "Maxwell"},
      {0x60, "Pascal"},
      {0x61, "Pascal"},
      {0x62, "Pascal"},
      {0x70, "Volta"},
      {0x72, "Xavier"},
      {0x75, "Turing"},
      {0x80, "Ampere"},
      {0x86, "Ampere"},
      {0x87, "Ampere"},
      {0x89, "Ada"},
      {0x90, "Hopper"},
      {0xa0, "Blackwell"},
      {0xa1, "Blackwell"},
      {0xc0, "Blackwell"},
      {-1, "Graphics Device"}};

  int index = 0;

  while (nGpuArchNameSM[index].SM != -1) {
    if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchNameSM[index].name;
    }

    index++;
  }

  printf(
      "MapSMtoArchName for SM %d.%d is undefined."
      "  Default to use %s\n",
      major, minor, nGpuArchNameSM[index - 1].name);
  return nGpuArchNameSM[index - 1].name;
}

inline int _ConvertSMVer2Cores(int major, int minor) {
  typedef struct {
    int SM;
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {0x87, 128},
      {0x89, 128},
      {0x90, 128},
      {0xa0, 128},
      {0xa1, 128},
      {0xc0, 128},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  printf(
      "MapSMtoCores for SM %d.%d is undefined."
      "  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

inline int gpuGetMaxGflopsDeviceId() {
  int current_device = 0, sm_per_multiproc = 0;
  int max_perf_device = 0;
  int device_count = 0;
  int devices_prohibited = 0;

  uint64_t max_compute_perf = 0;
  checkCudaErrors(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  current_device = 0;

  while (current_device < device_count) {
    int computeMode = -1, major = 0, minor = 0;
    checkCudaErrors(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, current_device));

    if (computeMode != hipComputeModeProhibited) {
      if (major == 9999 && minor == 9999) {
        sm_per_multiproc = 1;
      } else {
        sm_per_multiproc =
            _ConvertSMVer2Cores(major,  minor);
      }
      int multiProcessorCount = 0, clockRate = 0;
      checkCudaErrors(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, current_device));
      hipError_t result = hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, current_device);
      if (result != hipSuccess) {
        if(result == hipErrorInvalidValue) {
          clockRate = 1;
        }
        else {
          exit(EXIT_FAILURE);
        }
      }
      uint64_t compute_perf = (uint64_t)multiProcessorCount * sm_per_multiproc * clockRate;

      if (compute_perf > max_compute_perf) {
        max_compute_perf = compute_perf;
        max_perf_device = current_device;
      }
    } else {
      devices_prohibited++;
    }

    ++current_device;
  }

  if (devices_prohibited == device_count) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " all devices have compute mode prohibited.\n");
    exit(EXIT_FAILURE);
  }

  return max_perf_device;
}

inline int findCudaDevice(int argc, const char **argv) {
  int devID = 0;

  devID = gpuGetMaxGflopsDeviceId();
  checkCudaErrors(hipSetDevice(devID));
  int major = 0, minor = 0;
  checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
  checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
  printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
         devID, _ConvertSMVer2ArchName(major, minor), major, minor);

  return devID;
}

const char *sampleName = "simpleAssert";

bool testResult = true;

__global__ void testKernel(int N) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  assert(gtid < N);
}

void runTest(int argc, char **argv);

int main(int argc, char **argv) {
  printf("%s starting...\n", sampleName);

  runTest(argc, argv);

  printf("%s completed, returned %s\n", sampleName,
         testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void runTest(int argc, char **argv) {
  int Nblocks = 2;
  int Nthreads = 32;
  hipError_t error;

  findCudaDevice(argc, (const char **)argv);

  dim3 dimGrid(Nblocks);
  dim3 dimBlock(Nthreads);

  printf("Launch kernel to generate assertion failures\n");
  testKernel<<<dimGrid, dimBlock>>>(60);

  printf("\n-- Begin assert output\n\n");
  error = hipDeviceSynchronize();
  printf("\n-- End assert output\n\n");

  if (error == hipErrorAssert) {
    printf(
        "Device assert failed as expected, "
        "CUDA error message is: %s\n\n",
        hipGetErrorString(error));
  }

  testResult = error == hipErrorAssert;
}
