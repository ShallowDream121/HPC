#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

namespace cg = cooperative_groups;

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

inline const char* _ConvertSMVer2ArchName(int major, int minor) {
  typedef struct {
    int SM; 
    const char* name;
  } sSMtoArchName;

  sSMtoArchName nGpuArchNameSM[] = {
      {0x30, "Kepler"},
      {0x32, "Kepler"},
      {0x35, "Kepler"},
      {0x37, "Kepler"},
      {0x50, "Maxwell"},
      {0x52, "Maxwell"},
      {0x53, "Maxwell"},
      {0x60, "Pascal"},
      {0x61, "Pascal"},
      {0x62, "Pascal"},
      {0x70, "Volta"},
      {0x72, "Xavier"},
      {0x75, "Turing"},
      {0x80, "Ampere"},
      {0x86, "Ampere"},
      {0x87, "Ampere"},
      {0x89, "Ada"},
      {0x90, "Hopper"},
      {0xa0, "Blackwell"},
      {0xa1, "Blackwell"},
      {0xc0, "Blackwell"},
      {-1, "Graphics Device"}};

  int index = 0;

  while (nGpuArchNameSM[index].SM != -1) {
    if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchNameSM[index].name;
    }

    index++;
  }

  printf(
      "MapSMtoArchName for SM %d.%d is undefined."
      "  Default to use %s\n",
      major, minor, nGpuArchNameSM[index - 1].name);
  return nGpuArchNameSM[index - 1].name;
}

inline int _ConvertSMVer2Cores(int major, int minor) {
  typedef struct {
    int SM;
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60,  64},
      {0x61, 128},
      {0x62, 128},
      {0x70,  64},
      {0x72,  64},
      {0x75,  64},
      {0x80,  64},
      {0x86, 128},
      {0x87, 128},
      {0x89, 128},
      {0x90, 128},
      {0xa0, 128},
      {0xa1, 128},
      {0xc0, 128},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  printf(
      "MapSMtoCores for SM %d.%d is undefined."
      "  Default to use %d Cores/SM\n",
      major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores;
}

inline int gpuGetMaxGflopsDeviceId() {
  int current_device = 0, sm_per_multiproc = 0;
  int max_perf_device = 0;
  int device_count = 0;
  int devices_prohibited = 0;

  uint64_t max_compute_perf = 0;
  checkCudaErrors(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  current_device = 0;

  while (current_device < device_count) {
    int computeMode = -1, major = 0, minor = 0;
    checkCudaErrors(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, current_device));
    checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, current_device));

    if (computeMode != hipComputeModeProhibited) {
      if (major == 9999 && minor == 9999) {
        sm_per_multiproc = 1;
      } else {
        sm_per_multiproc =
            _ConvertSMVer2Cores(major,  minor);
      }
      int multiProcessorCount = 0, clockRate = 0;
      checkCudaErrors(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, current_device));
      hipError_t result = hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, current_device);
      if (result != hipSuccess) {
        if(result == hipErrorInvalidValue) {
          clockRate = 1;
        }
        else {
          exit(EXIT_FAILURE);
        }
      }
      uint64_t compute_perf = (uint64_t)multiProcessorCount * sm_per_multiproc * clockRate;

      if (compute_perf > max_compute_perf) {
        max_compute_perf = compute_perf;
        max_perf_device = current_device;
      }
    } else {
      devices_prohibited++;
    }

    ++current_device;
  }

  if (devices_prohibited == device_count) {
    fprintf(stderr,
            "gpuGetMaxGflopsDeviceId() CUDA error:"
            " all devices have compute mode prohibited.\n");
    exit(EXIT_FAILURE);
  }

  return max_perf_device;
}

inline int findCudaDevice(int argc, const char **argv) {
  int devID = 0;

  devID = gpuGetMaxGflopsDeviceId();
  checkCudaErrors(hipSetDevice(devID));
  int major = 0, minor = 0;
  checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devID));
  checkCudaErrors(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, devID));
  printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
         devID, _ConvertSMVer2ArchName(major, minor), major, minor);

  return devID;
}

const char *sSDKsample = "hyperQ";

__device__ void clock_block(clock_t *d_o, clock_t clock_count) {
  unsigned int start_clock = (unsigned int)clock();

  clock_t clock_offset = 0;

  while (clock_offset < clock_count) {
    unsigned int end_clock = (unsigned int)clock();
    clock_offset = (clock_t)(end_clock - start_clock);
  }

  d_o[0] = clock_offset;
}

__global__ void kernel_A(clock_t *d_o, clock_t clock_count) {
  clock_block(d_o, clock_count);
}
__global__ void kernel_B(clock_t *d_o, clock_t clock_count) {
  clock_block(d_o, clock_count);
}

__global__ void sum(clock_t *d_clocks, int N) {
  cg::thread_block cta = cg::this_thread_block();
  __shared__ clock_t s_clocks[32];

  clock_t my_sum = 0;

  for (int i = threadIdx.x; i < N; i += blockDim.x) {
    my_sum += d_clocks[i];
  }

  s_clocks[threadIdx.x] = my_sum;
  cg::sync(cta);

  for (int i = warpSize / 2; i > 0; i /= 2) {
    if (threadIdx.x < i) {
      s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
    }

    cg::sync(cta);
  }

  if (threadIdx.x == 0) {
    d_clocks[0] = s_clocks[0];
  }
}

int main(int argc, char **argv) {
  int nstreams = 32;       
  float kernel_time = 10;  
  float elapsed_time;
  int cuda_device = 0;

  printf("starting %s...\n", sSDKsample);

  cuda_device = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDevice(&cuda_device));
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

  if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5)) {
    if (deviceProp.concurrentKernels == 0) {
      printf(
          "> GPU does not support concurrent kernel execution (SM 3.5 or "
          "higher required)\n");
      printf("  CUDA kernel runs will be serialized\n");
    } else {
      printf("> GPU does not support HyperQ\n");
      printf("  CUDA kernel runs will have limited concurrency\n");
    }
  }

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

  clock_t *a = 0;
  checkCudaErrors(hipHostMalloc((void **)&a, sizeof(clock_t)));

  clock_t *d_a = 0;
  checkCudaErrors(hipMalloc((void **)&d_a, 2 * nstreams * sizeof(clock_t)));

  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  for (int i = 0; i < nstreams; i++) {
    checkCudaErrors(hipStreamCreate(&(streams[i])));
  }

  hipEvent_t start_event, stop_event;
  checkCudaErrors(hipEventCreate(&start_event));
  checkCudaErrors(hipEventCreate(&stop_event));

#if defined(__arm__) || defined(__aarch64__)
  clock_t time_clocks = (clock_t)(kernel_time * (deviceProp.clockRate / 100));
#else
  clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
#endif
  clock_t total_clocks = 0;

  checkCudaErrors(hipEventRecord(start_event, 0));

  for (int i = 0; i < nstreams; ++i) {
    kernel_A<<<1, 1, 0, streams[i]>>>(&d_a[2 * i], time_clocks);
    total_clocks += time_clocks;
    kernel_B<<<1, 1, 0, streams[i]>>>(&d_a[2 * i + 1], time_clocks);
    total_clocks += time_clocks;
  }

  checkCudaErrors(hipEventRecord(stop_event, 0));

  sum<<<1, 32>>>(d_a, 2 * nstreams);
  checkCudaErrors(hipMemcpy(a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost));

  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

  printf(
      "Expected time for serial execution of %d sets of kernels is between "
      "approx. %.3fs and %.3fs\n",
      nstreams, (nstreams + 1) * kernel_time / 1000.0f,
      2 * nstreams * kernel_time / 1000.0f);
  printf(
      "Expected time for fully concurrent execution of %d sets of kernels is "
      "approx. %.3fs\n",
      nstreams, 2 * kernel_time / 1000.0f);
  printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

  bool bTestResult = (a[0] >= total_clocks);

  for (int i = 0; i < nstreams; i++) {
    hipStreamDestroy(streams[i]);
  }

  free(streams);
  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);
  hipHostFree(a);
  hipFree(d_a);

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
